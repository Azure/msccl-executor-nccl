#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2015-2021, NVIDIA CORPORATION. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

#include "devcomm.h"
#include "collectives.h"
#include "common.h"

__shared__ ncclShmemData ncclShmem;
#if __CUDA_ARCH__ < 700
  __shared__ ulong2 ncclShmemPerWarp[ncclShmemScratchWarpSize()*(NCCL_MAX_NTHREADS/WARP_SIZE)/sizeof(ulong2)];
#endif

#define NCCL_FUNC5(func, algo, devredop, type, nullify) \
  MACRO_IF(nullify, nullptr, NCCL_FUNC_NAME(func, algo, LL,     devredop, type)), \
  MACRO_IF(nullify, nullptr, NCCL_FUNC_NAME(func, algo, LL128,  devredop, type)), \
  MACRO_IF(nullify, nullptr, NCCL_FUNC_NAME(func, algo, SIMPLE, devredop, type))

#define NCCL_FUNC4(func, devredop, type, nullify) \
  NCCL_FUNC5(func, TREE,    devredop, type, nullify), \
  NCCL_FUNC5(func, RING,    devredop, type, nullify), \
  NCCL_FUNC5(func, COLLNET_DIRECT, devredop, type, nullify), \
  NCCL_FUNC5(func, COLLNET_CHAIN,  devredop, type, nullify), \
  NCCL_FUNC5(func, NVLS,           devredop, type, nullify)

#if defined(__CUDA_BF16_TYPES_EXIST__)
// Must be consistent with ncclDataType_t
#define NCCL_FUNCS3A(func, devredop, nullForFloat) \
  NCCL_FUNC4(func, devredop, int8_t, 0), \
  NCCL_FUNC4(func, devredop, uint8_t, 0), \
  NCCL_FUNC4(func, devredop, int32_t, 0), \
  NCCL_FUNC4(func, devredop, uint32_t, 0), \
  NCCL_FUNC4(func, devredop, int64_t, 0), \
  NCCL_FUNC4(func, devredop, uint64_t, 0), \
  NCCL_FUNC4(func, devredop, half, nullForFloat), \
  NCCL_FUNC4(func, devredop, float, nullForFloat), \
  NCCL_FUNC4(func, devredop, double, nullForFloat), \
  NCCL_FUNC4(func, devredop, __hip_bfloat16, nullForFloat), \
  NCCL_FUNC4(func, devredop, __hip_fp8_e4m3_fnuz, nullForFloat), \
  NCCL_FUNC4(func, devredop, __hip_fp8_e5m2_fnuz, nullForFloat)
#define NCCL_FUNCS3B(func, devredop) \
  NCCL_FUNC4(func, devredop, int8_t, 0), \
  NCCL_FUNC4(func, devredop, int8_t, 0), \
  NCCL_FUNC4(func, devredop, int8_t, 0), \
  NCCL_FUNC4(func, devredop, int8_t, 0), \
  NCCL_FUNC4(func, devredop, int8_t, 0), \
  NCCL_FUNC4(func, devredop, int8_t, 0), \
  NCCL_FUNC4(func, devredop, int8_t, 0), \
  NCCL_FUNC4(func, devredop, int8_t, 0), \
  NCCL_FUNC4(func, devredop, int8_t, 0), \
  NCCL_FUNC4(func, devredop, int8_t, 0), \
  NCCL_FUNC4(func, devredop, int8_t, 0), \
  NCCL_FUNC4(func, devredop, int8_t, 0)
#else
// Must be consistent with ncclDataType_t
#define NCCL_FUNCS3A(func, devredop, nullForFloat) \
  NCCL_FUNC4(func, devredop, int8_t, 0), \
  NCCL_FUNC4(func, devredop, uint8_t, 0), \
  NCCL_FUNC4(func, devredop, int32_t, 0), \
  NCCL_FUNC4(func, devredop, uint32_t, 0), \
  NCCL_FUNC4(func, devredop, int64_t, 0), \
  NCCL_FUNC4(func, devredop, uint64_t, 0), \
  NCCL_FUNC4(func, devredop, half, nullForFloat), \
  NCCL_FUNC4(func, devredop, float, nullForFloat), \
  NCCL_FUNC4(func, devredop, double, nullForFloat)
#define NCCL_FUNCS3B(func, devredop) \
  NCCL_FUNC4(func, devredop, int8_t, 0), \
  NCCL_FUNC4(func, devredop, int8_t, 0), \
  NCCL_FUNC4(func, devredop, int8_t, 0), \
  NCCL_FUNC4(func, devredop, int8_t, 0), \
  NCCL_FUNC4(func, devredop, int8_t, 0), \
  NCCL_FUNC4(func, devredop, int8_t, 0), \
  NCCL_FUNC4(func, devredop, int8_t, 0), \
  NCCL_FUNC4(func, devredop, int8_t, 0), \
  NCCL_FUNC4(func, devredop, int8_t, 0)
#endif

// Must be consistent with ncclRedOp_t
#define NCCL_FUNCS2A(func) \
  NCCL_FUNCS3A(func, Sum,        /*nullForFloat=*/0), \
  NCCL_FUNCS3A(func, Prod,       /*nullForFloat=*/0), \
  NCCL_FUNCS3A(func, Max,        /*nullForFloat=*/0), \
  NCCL_FUNCS3A(func, Min,        /*nullForFloat=*/0), \
  NCCL_FUNCS3A(func, PreMulSum,  /*nullForFloat=*/0), \
  NCCL_FUNCS3A(func, SumPostDiv, /*nullForFloat=*/1)

#define NCCL_FUNCS2B(func) \
  NCCL_FUNCS3B(func, Sum), \
  NCCL_FUNCS3B(func, Sum), \
  NCCL_FUNCS3B(func, Sum), \
  NCCL_FUNCS3B(func, Sum), \
  NCCL_FUNCS3B(func, Sum), \
  NCCL_FUNCS3B(func, Sum)

// Must be consistent with the ncclFuncSet enum
__device__ ncclKern_t ncclFuncs[1+ncclNumTypes+NCCL_NUM_FUNCTIONS*ncclNumDevRedOps*ncclNumTypes*NCCL_NUM_ALGORITHMS*NCCL_NUM_PROTOCOLS] = {
// Don't try to initialize the host shadow copy of this device-side global
// variable. There is no host pointer to a device-side function, which
// confuses clang. This will be fixed in the next clang release.
#if __CUDA_ARCH__
  NCCL_FUNC_NAME(SendRecv, RING, SIMPLE, Sum, int8_t),
  NCCL_ONERANK_REDUCE_NAME(PreMulSum, int8_t),
  NCCL_ONERANK_REDUCE_NAME(PreMulSum, uint8_t),
  NCCL_ONERANK_REDUCE_NAME(PreMulSum, int32_t),
  NCCL_ONERANK_REDUCE_NAME(PreMulSum, uint32_t),
  NCCL_ONERANK_REDUCE_NAME(PreMulSum, int64_t),
  NCCL_ONERANK_REDUCE_NAME(PreMulSum, uint64_t),
  NCCL_ONERANK_REDUCE_NAME(PreMulSum, half),
  NCCL_ONERANK_REDUCE_NAME(PreMulSum, float),
  NCCL_ONERANK_REDUCE_NAME(PreMulSum, double),
  #if defined(__CUDA_BF16_TYPES_EXIST__)
    NCCL_ONERANK_REDUCE_NAME(PreMulSum, __hip_bfloat16),
    NCCL_ONERANK_REDUCE_NAME(PreMulSum, __hip_fp8_e4m3_fnuz),
    NCCL_ONERANK_REDUCE_NAME(PreMulSum, __hip_fp8_e5m2_fnuz),
  #endif
  NCCL_FUNCS2B(Broadcast),
  NCCL_FUNCS2A(Reduce),
  NCCL_FUNCS2B(AllGather),
  NCCL_FUNCS2A(ReduceScatter),
  NCCL_FUNCS2A(AllReduce)
#endif
};

// Workaround for https://reviews.llvm.org/D55580
__device__ void ncclWorkaroundClangD55580() {}
